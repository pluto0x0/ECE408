#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 32

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  #define Matrix(M, r, c) (M[num##M##Columns * (r) + (c)])

  __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
  int row = TILE_WIDTH * by + ty;
  int col = TILE_WIDTH * bx + tx;
  Matrix(C, row, col) = 0.;
  float Cvalue = 0.;
  for(int q = 0; q < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; q++) {    // 0 <= q < ceil(numAColumns / TILEWIDTH)
    int orgRow = by * TILE_WIDTH + ty, orgCol = q * TILE_WIDTH + tx;
    tileA[ty][tx] = (orgRow < numARows && orgCol < numAColumns) ? Matrix(A, orgRow, orgCol) : 0.;
    orgRow = q * TILE_WIDTH + ty, orgCol = bx * TILE_WIDTH + tx;
    tileB[ty][tx] = (orgRow < numBRows && orgCol < numBColumns) ? Matrix(B, orgRow, orgCol) : 0.;
    __syncthreads();    // q-th tile loaded
    for(int i = 0; i < TILE_WIDTH; i++)
    Cvalue += tileA[ty][i] * tileB[i][tx];
    __syncthreads();
  }
  if(row < numCRows && col < numCColumns)
    Matrix(C, row, col) = Cvalue;
  #undef Matrix
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc(&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc(&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc(&deviceC, numCRows * numCColumns * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(numCColumns * 1. / TILE_WIDTH), ceil(numCRows * 1. / TILE_WIDTH), 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);       // 32 * 32 = 1024

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC,
                                        numARows, numAColumns,
                                        numBRows, numBColumns,
                                        numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
