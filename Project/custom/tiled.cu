
#include <hip/hip_runtime.h>
#ifdef TILED_CU
#include <cmath>
#include <iostream>

#include "gpu-new-forward.h"

#define BLOCK_SIZE 18

#define CHECK_ERR                                                              \
    {                                                                          \
        cudaError_t error = cudaGetLastError();                                \
        if (error != cudaSuccess) {                                            \
            std::cerr << "CUDA error: " << __FILE__ << ":" << __LINE__ << ": " \
                      << cudaGetErrorString(error) << std::endl;               \
            exit(-1);                                                          \
        }                                                                      \
    }

#define CONST_SIZE 6000
__constant__ float mask[CONST_SIZE];

#define TILE_WIDTH ((BLOCK_SIZE - 1) * S + K)

__global__ void conv_forward_kernel(float *output, const float *input, const int B,
                                    const int M, const int C, const int H, const int W,
                                    const int K, const int S) {
    /*
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K) / S + 1;
    const int W_out = (W - K) / S + 1;

#define out_4d(i3, i2, i1, i0) \
    output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define mask_4d(i3, i2, i1, i0) \
    mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

#define in_2d(i1, i0) Input[(i1) * (W) + i0]
#define mask_2d(i1, i0) Mask[(i1) * (K) + i0]

    const unsigned int b = blockIdx.x;
    unsigned int W_blksize =
        (W_out - 1) / BLOCK_SIZE + 1;  // number of horizontal tiles per output map
    unsigned int m = blockIdx.y;       // output channel
    unsigned int h_s = (blockIdx.z / W_blksize) * BLOCK_SIZE + threadIdx.y;
    unsigned int h = h_s * S;
    unsigned int w_s = (blockIdx.z % W_blksize) * BLOCK_SIZE + threadIdx.x;
    unsigned int w = w_s * S;

    // if (w + K - 1 >= W || h + K - 1 >= H) return;

    float acc = 0.;
#pragma unroll
    for (int c = 0; c < C; c++) {  // sum over all input channels
#pragma unroll
        for (int p = 0; p < K; p++) {  // loop over KxK filter
#pragma unroll
            for (int q = 0; q < K; q++)
                acc += in_4d(b, c, h + q, w + p) * mask_4d(m, c, q, p);
        }
    }
    if (w + K - 1 < W && h + K - 1 < H) out_4d(b, m, h_s, w_s) = acc;

#undef out_4d
#undef in_4d
#undef mask_4d
}

__global__ void conv_forward_kernel_S1(float *__restrict__ output,
                                       const float *__restrict__ input, const int B,
                                       const int M, const int C, const int H, const int W,
                                       const int K, const int S) {
    /*
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K) + 1;
    const int W_out = (W - K) + 1;

#define out_4d(i3, i2, i1, i0) \
    output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define mask_4d(i3, i2, i1, i0) \
    mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
#define tile(i2, i1, i0) tile_input[(i2) * tw * tw + tw * (i1) + (i0)]

    extern __shared__ float tile_input[];

    const unsigned int b = blockIdx.x;
    unsigned int m = blockIdx.y;  // output channel
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int W_blksize =
        (W_out - 1) / BLOCK_SIZE + 1;  // number of horizontal tiles per output map
    unsigned int h = (blockIdx.z / W_blksize) * BLOCK_SIZE + ty;
    unsigned int w = (blockIdx.z % W_blksize) * BLOCK_SIZE + tx;
    const unsigned int tw = TILE_WIDTH;

    float acc = 0.;

    if (h < H && w < W) {
#pragma unroll
        for (int c = 0; c < C; c++) {
            tile(c, ty, tx) = in_4d(b, c, h, w);
        }
    }
    __syncthreads();
    if (tx < BLOCK_SIZE && ty < BLOCK_SIZE)
#pragma unroll
        for (int c = 0; c < C; c++) {
// if(w + K - 1 < W && h + K - 1 < H)
#pragma unroll
            for (int x = 0; x < K; x++)
#pragma unroll
                for (int y = 0; y < K; y++)
                    acc += tile(c, ty + y, tx + x) * mask_4d(m, c, y, x);
        }
    if (tx < BLOCK_SIZE && ty < BLOCK_SIZE)
        if (h < H_out && w < W_out) out_4d(b, m, h, w) = acc;

#undef tile
#undef out_4d
#undef in_4d
#undef mask_4d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(
    const float *host_output, const float *host_input, const float *host_mask,
    float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr,
    const int B, const int M, const int C, const int H, const int W, const int K,
    const int S) {
    unsigned int H_out = (H - K) / S + 1;
    unsigned int W_out = (W - K) / S + 1;
    unsigned int size_in = B * W * H * C * sizeof(float);
    unsigned int size_out = B * H_out * W_out * M * sizeof(float);
    unsigned int size_mask = K * K * M * C * sizeof(float);

    cudaMalloc(device_input_ptr, size_in);
    cudaMalloc(device_output_ptr, size_out);
    // cudaMalloc(device_mask_ptr, size_mask);

    cudaMemcpy(*device_input_ptr, host_input, size_in, cudaMemcpyHostToDevice);
    cudaMemcpy(*device_output_ptr, host_output, size_out, cudaMemcpyHostToDevice);
    // cudaMemcpy(*device_mask_ptr, host_mask, size_mask, cudaMemcpyHostToDevice);

    cudaMemcpyToSymbol(mask, host_mask, size_mask);

    CHECK_ERR;
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output,
                                             const float *device_input,
                                             const float *device_mask, const int B,
                                             const int M, const int C, const int H,
                                             const int W, const int K, const int S) {
    std::cout << "Running in " << __FILE__ << std::endl;
    std::cout << "K = " << K << std::endl;
    std::cout << "S = " << S << std::endl;

    unsigned int H_out = (H - K) / S + 1;
    unsigned int W_out = (W - K) / S + 1;
    unsigned int W_blksize =
        (W_out - 1) / BLOCK_SIZE + 1;  // number of horizontal tiles per output map
    unsigned int H_blksize =
        (H_out - 1) / BLOCK_SIZE + 1;        // number of vertical tiles per output map
    unsigned int Y = H_blksize * W_blksize;  // total number of tiles per map

    if (S > 1) {
        dim3 grid(B, M, Y);
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);  // output tile for untiled code
        conv_forward_kernel<<<grid, block>>>(device_output, device_input, B, M, C, H, W,
                                             K, S);
    } else {
        dim3 grid(B, M, Y);
        dim3 block(TILE_WIDTH,
                   TILE_WIDTH);  // output tile for untiled code
        unsigned int shared_size = C * TILE_WIDTH * TILE_WIDTH * sizeof(float);
        conv_forward_kernel_S1<<<grid, block, shared_size>>>(device_output, device_input,
                                                             B, M, C, H, W, K, S);
    }

    CHECK_ERR;
}

__host__ void GPUInterface::conv_forward_gpu_epilog(
    float *host_output, float *device_output, float *device_input, float *device_mask,
    const int B, const int M, const int C, const int H, const int W, const int K,
    const int S) {
    // Copy the output back to host
    unsigned int H_out = (H - K) / S + 1;
    unsigned int W_out = (W - K) / S + 1;
    unsigned int size_out = B * H_out * W_out * M * sizeof(float);
    cudaMemcpy(host_output, device_output, size_out, cudaMemcpyDeviceToHost);

    // Free device memory
    cudaFree(device_input);
    cudaFree(device_output);
    // cudaFree(device_mask);

    CHECK_ERR;
}

__host__ void GPUInterface::get_device_properties() {
    int deviceCount;
    cudaGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        cudaDeviceProp deviceProp;
        cudaGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
        std::cout << "Computational capabilities: " << deviceProp.major << "."
                  << deviceProp.minor << std::endl;
        std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
        std::cout << "Max Constant memory size: " << deviceProp.totalConstMem
                  << std::endl;
        std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock
                  << std::endl;
        std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock
                  << std::endl;
        std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, "
                  << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2]
                  << " z" << std::endl;
        std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, "
                  << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2]
                  << " z" << std::endl;
        std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
    }
}

#endif